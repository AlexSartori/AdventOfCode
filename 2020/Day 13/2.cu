#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define MAX_IDS 1000
#define CUDA_BLOCKS 30
#define CUDA_THREADS 32
#define CUDA_STRIDE 1e9L

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"\nGPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__
void cuda_work(int* IDs, int nIDs, long long start, long long increment, int offset, long long* solution) {
  long long int start_t = start + (blockIdx.x * blockDim.x + threadIdx.x) * CUDA_STRIDE;
  long long int end_t = start_t + CUDA_STRIDE;
  long long t = start_t - (start_t % increment);

  while (*solution == -1 && t < end_t) {
	if (t == 0)
	  t = increment;

	int i;
	for (i = 0; i < nIDs; i++)
      if (IDs[i] != 0 && (t-offset+i) % IDs[i] != 0)
        break;

	if (i == nIDs) {
      *solution = t - offset;
	  printf("\n\n!!! Thr. %dx%d found solution: %ld\n", threadIdx.x, blockIdx.x, *solution);
	}

    t += increment;
  }
}


int get_next_id(FILE *fp) {
  char id[20];
  int len = 0;

  while ((id[len]=fgetc(fp)) != ',' && id[len] != EOF)
    len++;
  id[len] = '\0';

  return len == 0 ? -1 : atoi(id);
}

int main() {
  int *IDs = (int*)malloc(sizeof(int)*MAX_IDS);
  int nIDs = 0;
  int increment = 1, offset = 0;
  long long *solution = (long long*)malloc(sizeof(long long));
  *solution = -1;

  FILE *f = fopen("input", "r");
  while (fgetc(f) != '\n') ;
  for (nIDs = 0; (IDs[nIDs] = get_next_id(f)) != -1; nIDs++)
    if (IDs[nIDs] - nIDs > increment) {
      increment = IDs[nIDs];
	  offset = nIDs;
	}

  int *d_IDs;
  long long *d_solution;
  gpuErrchk(hipMalloc(&d_IDs, sizeof(int)*nIDs));
  gpuErrchk(hipMalloc(&d_solution, sizeof(long long)));
  gpuErrchk(hipMemcpy(d_IDs, IDs, sizeof(int)*nIDs, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_solution, solution, sizeof(long long), hipMemcpyHostToDevice));


  long long int workload = CUDA_BLOCKS * CUDA_THREADS * CUDA_STRIDE;
  for (int stride_idx = 0; *solution == -1 && stride_idx < 1e15L/workload; stride_idx++) {
    long long start = workload * stride_idx;
    printf("Stride %d @ %ld B - c.a. %ld%\r", stride_idx, (long long)(start/1e9L), (long long)(start/1e13L));

	  cuda_work<<<CUDA_BLOCKS, CUDA_THREADS>>>(d_IDs, nIDs, start, increment, offset, d_solution);
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(solution, d_solution, sizeof(long long), hipMemcpyDeviceToHost));
  }

  printf("\n\n>>> Solution: %ld\n", *solution);

  hipFree(IDs);
  hipFree(d_solution);
  free(IDs);
  free(solution);
}
